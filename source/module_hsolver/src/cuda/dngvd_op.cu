#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"
#include "src_pdiag/hip/hip_runtime_api.h"

#include <hipsolver.h>

#define cusolverErrcheck(res)                      \
    {                                              \
        cusolverAssert((res), __FILE__, __LINE__); \
    }

// cuSOLVER API errors
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

inline void cusolverAssert(hipsolverStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSOLVER Assert: %s %s %d\n", _cusolverGetErrorEnum(code), file, line);
        if (abort)
            exit(code);
    }
}

namespace hsolver
{

static hipsolverHandle_t cusolver_H = nullptr;

void createCUSOLVERhandle()
{
    if (cusolver_H == nullptr)
    {
        cusolverErrcheck(hipsolverDnCreate(&cusolver_H));
    }
}

void destoryCUSOLVERhandle()
{
    if (cusolver_H != nullptr)
    {
        cusolverErrcheck(hipsolverDnDestroy(cusolver_H));
        cusolver_H = nullptr;
    }
}

template <>
void dngvx_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                   const int nstart,
                                                   const int ldh,
                                                   const std::complex<double>* A,
                                                   const std::complex<double>* B,
                                                   const int m,
                                                   double* W,
                                                   std::complex<double>* V)
{
    // init A_eigenvectors, transpose_B and all_W
    double2 *A_eigenvectors, *transpose_B;
    if (nstart == ldh)
    {
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * nstart * nstart));
        checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * nstart * nstart));

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, A, (std::complex<double>*)A_eigenvectors);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, B, (std::complex<double>*)transpose_B);
    }
    else if (nstart < ldh)
    {
        // nstart < ldh
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * nstart * nstart));
        checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * nstart * nstart));

        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, nstart, A, ldh, (std::complex<double>*)A_eigenvectors, nstart);
        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, nstart, B, ldh, (std::complex<double>*)transpose_B, nstart);

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d,
                                                      nstart,
                                                      nstart,
                                                      (std::complex<double>*)A_eigenvectors,
                                                      (std::complex<double>*)A_eigenvectors);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d,
                                                      nstart,
                                                      nstart,
                                                      (std::complex<double>*)transpose_B,
                                                      (std::complex<double>*)transpose_B);
    }
    else if (nstart > ldh)
    {
        assert(nstart < ldh);
    }

    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * nstart));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        nstart,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        nstart,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get eigenvalues and eigenvectors.  only m !
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * m, hipMemcpyDeviceToDevice));

    if (ldh == nstart)
    {
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, V, V);
        checkCudaErrors(
            hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>) * nstart * m, hipMemcpyDeviceToDevice));
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, V, V);
    }
    else
    {
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, ldh, V, V);
        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, m, (std::complex<double>*)A_eigenvectors, nstart, V, ldh);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, ldh, V, V);
    }

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(transpose_B));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

template <>
void dngv_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                  const int nstart,
                                                  const int ldh,
                                                  const std::complex<double>* A,
                                                  const std::complex<double>* B,
                                                  double* W,
                                                  std::complex<double>* V)
{
    assert(nstart == ldh);
    // init A_eigenvectors & transpose_B
    double2 *A_eigenvectors, *transpose_B;
    checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * ldh * nstart));
    checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * ldh * nstart));

    // transpose A, B  to A_eigenvectors, transpose_B
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, B, (std::complex<double>*)transpose_B);

    // init all_W
    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * ldh));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get all eigenvalues and eigenvectors.
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * ldh, hipMemcpyDeviceToDevice));
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, (std::complex<double>*)A_eigenvectors, V);

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(transpose_B));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

template <>
void dngvd_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                   const int nstart,
                                                   const int ldh,
                                                   const std::complex<double>* A,
                                                   const std::complex<double>* B,
                                                   double* W,
                                                   std::complex<double>* V)
{
    assert(nstart == ldh);
    // init A_eigenvectors & transpose_B
    double2 *A_eigenvectors, *transpose_B;
    checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * ldh * nstart));
    checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * ldh * nstart));

    // transpose A, B  to A_eigenvectors, transpose_B
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, B, (std::complex<double>*)transpose_B);

    // init all_W
    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * ldh));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get all eigenvalues and eigenvectors.
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * ldh, hipMemcpyDeviceToDevice));
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, (std::complex<double>*)A_eigenvectors, V);

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(transpose_B));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

template <>
void dnevx_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                   const int nstart,
                                                   const int ldh,
                                                   const std::complex<double>* A,
                                                   const int m,
                                                   double* W,
                                                   std::complex<double>* V)
{
    // init A_eigenvectors, transpose_B and all_W
    double2 *A_eigenvectors;
    if (nstart == ldh)
    {
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * nstart * nstart));

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, A, (std::complex<double>*)A_eigenvectors);
    }
    else if (nstart < ldh)
    {
        // nstart < ldh
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * nstart * nstart));

        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, nstart, A, ldh, (std::complex<double>*)A_eigenvectors, nstart);

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d,
                                                      nstart,
                                                      nstart,
                                                      (std::complex<double>*)A_eigenvectors,
                                                      (std::complex<double>*)A_eigenvectors);
    }
    else if (nstart > ldh)
    {
        assert(nstart < ldh);
    }

    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * nstart));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZheevd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        nstart,
        A_eigenvectors,
        nstart,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZheevd(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        nstart,
        A_eigenvectors,
        nstart,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get eigenvalues and eigenvectors.  only m !
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * m, hipMemcpyDeviceToDevice));

    if (ldh == nstart)
    {
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, V, V);
        checkCudaErrors(
            hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>) * nstart * m, hipMemcpyDeviceToDevice));
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, nstart, nstart, V, V);
    }
    else
    {
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, ldh, V, V);
        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, m, (std::complex<double>*)A_eigenvectors, nstart, V, ldh);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, ldh, V, V);
    }

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

} // namespace hsolver