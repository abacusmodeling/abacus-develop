#include "hip/hip_runtime.h"
#include "module_pw/include/pw_multi_device.h"
#include "thrust/complex.h"
#include <hip/hip_runtime.h>

namespace ModulePW{

#define THREADS_PER_BLOCK 256

template<class FPTYPE>
__global__ void set_3d_fft_box(
    const int npwk,
    const int* box_index,
    const thrust::complex<FPTYPE>* in,
    thrust::complex<FPTYPE>* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < npwk)
    {
        int xx = box_index[idx];
        out[xx] = in[idx];
    }
}

template<class FPTYPE>
__global__ void set_recip_to_real_output(
    const int nrxx,
    const bool add,
    const FPTYPE factor,
    const thrust::complex<FPTYPE>* in,
    thrust::complex<FPTYPE>* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= nrxx) {return;}
    if(add) {
        out[idx] += factor * in[idx];
    }
    else {
        out[idx] = in[idx];
    }
}

template<class FPTYPE>
__global__ void set_real_to_recip_output(
    const int npwk,
    const int nxyz,
    const bool add,
    const FPTYPE factor,
    const int* box_index,
    const thrust::complex<FPTYPE>* in,
    thrust::complex<FPTYPE>* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= npwk) {return;}
    if(add) {
        out[idx] += factor / nxyz * in[box_index[idx]];
    }
    else {
        out[idx] = in[box_index[idx]] / nxyz;
    }
}

template <typename FPTYPE>
void set_3d_fft_box_op<FPTYPE, psi::DEVICE_GPU>::operator()(
    const psi::DEVICE_GPU*  /*dev*/,
    const int npwk,
    const int* box_index,
    const std::complex<FPTYPE>* in,
    std::complex<FPTYPE>* out)
{
    const int block = (npwk + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    set_3d_fft_box<double><<<block, THREADS_PER_BLOCK>>>(
        npwk,
        box_index,
        reinterpret_cast<const thrust::complex<FPTYPE>*>(in),
        reinterpret_cast<thrust::complex<FPTYPE>*>(out));
}

template <typename FPTYPE>
void set_recip_to_real_output_op<FPTYPE, psi::DEVICE_GPU>::operator()(
    const psi::DEVICE_GPU*  /*dev*/,
    const int nrxx,
    const bool add,
    const FPTYPE factor,
    const std::complex<FPTYPE>* in,
    std::complex<FPTYPE>* out)
{
    const int block = (nrxx + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    set_recip_to_real_output<double><<<block, THREADS_PER_BLOCK>>>(
        nrxx,
        add,
        factor,
        reinterpret_cast<const thrust::complex<FPTYPE>*>(in),
        reinterpret_cast<thrust::complex<FPTYPE>*>(out));
}

template <typename FPTYPE>
void set_real_to_recip_output_op<FPTYPE, psi::DEVICE_GPU>::operator()(
    const psi::DEVICE_GPU*  /*dev*/,
    const int npwk,
    const int nxyz,
    const bool add,
    const FPTYPE factor,
    const int* box_index,
    const std::complex<FPTYPE>* in,
    std::complex<FPTYPE>* out)
{
    const int block = (npwk + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    set_real_to_recip_output<double><<<block, THREADS_PER_BLOCK>>>(
        npwk,
        nxyz,
        add,
        factor,
        box_index,
        reinterpret_cast<const thrust::complex<FPTYPE>*>(in),
        reinterpret_cast<thrust::complex<FPTYPE>*>(out));
}

template struct set_3d_fft_box_op<double, psi::DEVICE_GPU>;
template struct set_recip_to_real_output_op<double, psi::DEVICE_GPU>;
template struct set_real_to_recip_output_op<double, psi::DEVICE_GPU>;

}  // namespace ModulePW

