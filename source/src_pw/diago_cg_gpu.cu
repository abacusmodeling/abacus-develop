#include "hip/hip_runtime.h"
#include "diago_cg_gpu.h"
#include "hip/hip_runtime.h"
#include "global.h"

template<class T, class T2>
int Diago_CG_GPU<T, T2>::moved = 0;

template<class T, class T2>
__global__ void kernel_normalization(T2 *data, int size, T norm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

template<class T, class T2>
__global__ void kernel_precondition(T2 *res, const T2 *data, const int size, const T *P)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        res[idx].x = data[idx].x / P[idx];
        res[idx].y = data[idx].y / P[idx];
    }
}

template<class T, class T2>
__global__ void kernel_precondition_inverse(T2 *res, const T2 *data, const int size, const T *P)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        res[idx].x = data[idx].x * P[idx];
        res[idx].y = data[idx].y * P[idx];
    }
}

template<class T, class T2>
__global__ void kernel_get_gredient(T2 *g, T2 *ppsi, int size, T lambda)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        g[idx].x -= lambda * ppsi[idx].x;
        g[idx].y -= lambda * ppsi[idx].y;
    }
}

template<class T, class T2>
__global__ void kernel_get_gammacg(int size, T2 *dst, const T2 *src, T gamma)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x = dst[idx].x * gamma + src[idx].x;
        dst[idx].y = dst[idx].y * gamma + src[idx].y;
    }
}

template<class T, class T2>
__global__ void kernel_get_normacg(int size, T2 *dst, const T2 *src, T norma)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x = dst[idx].x - norma * src[idx].x;
        dst[idx].y = dst[idx].y - norma * src[idx].y;
    }
}

template<class T, class T2>
__global__ void kernel_multi_add(T2 *dst, T2 *src1, T a1, const T2 *src2, T a2, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x = src1[idx].x * a1 + src2[idx].x * a2;
        dst[idx].y = src1[idx].y * a1 + src2[idx].y * a2;
    }
}

template<class T, class T2>
Diago_CG_GPU<T, T2>::Diago_CG_GPU()
{
    test_cg=0;
    hipblasCreate(&diag_handle);
    // hipblasCreate(&ddot_handle);
}

template<class T, class T2>
Diago_CG_GPU<T, T2>::~Diago_CG_GPU() 
{
    hipblasDestroy(diag_handle);
    // hipblasDestroy(ddot_handle);
}

template<class T, class T2>
void Diago_CG_GPU<T, T2>::diag
(
    T2 *phi, // matrix nband*dim
    T *e,
    const int &dim,
    const int &dmx,
    const int &n_band,
    const T *precondition,
    const T &eps,
    const int &maxter,
    const bool &reorder,
    int &notconv,
    T &avg_iter
)
{

    // cout<<"begin diago fft dim"<<GlobalC::pw.nx<<" "<<GlobalC::pw.ny<<" "<<GlobalC::pw.nz<<endl;
    // cout << &GlobalC::pw << endl;
    if (test_cg==1) ModuleBase::TITLE("Diago_CG_GPU","diag");
    ModuleBase::timer::tick("Diago_CG_GPU","diag");

    avg_iter = 0.0;
    notconv = 0;
    // ZEROS(e, n_band);

    //-------------------------------------------------------------------
    // "poor man" iterative diagonalization of a complex hermitian matrix
    // through preconditioned conjugate gradient algorithm
    // Band-by-band algorithm with minimal use of memory
    // Calls h_1phi and s_1phi to calculate H|phi> and S|phi>
    // Works for generalized eigenvalue problem (US pseudopotentials) as well
    //-------------------------------------------------------------------

    T2 *sphi;
    T2 *scg;
    T2 *hphi;
    T2 *g;
    T2 *cg;
    T2 *g0;
    T2 *pphi;
    T2 *lagrange;
    T2 *phi_m;

    // cout << "Hello, CG!" << endl;
    // cout << "CG Dim = " << dim << " & " << dmx << endl;

    CHECK_CUDA(hipMalloc((void**)&sphi, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&scg, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&hphi, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&g, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&cg, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&g0, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&pphi, dim * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&lagrange, n_band * sizeof(T2)));
    CHECK_CUDA(hipMalloc((void**)&phi_m, dim * sizeof(T2)));

    // timer::tick("Diago_CG_GPU","diag");

	// Init with ZERO ...
    // T em_host = 0;

    for (int m=0; m<n_band; m++)
    {
        if (test_cg>2) GlobalV::ofs_running << "Diagonal Band : " << m << endl;

        CHECK_CUDA(hipMemcpy(phi_m, &phi[m*dmx], dim*sizeof(T2), hipMemcpyDeviceToDevice));

        // CHECK_CUDA(hipMemcpy(sphi, phi_m, dim * sizeof(T2), hipMemcpyDeviceToDevice));
        GlobalC::hm.hpw.s_1psi_gpu(dim, phi_m, sphi);

        this->schmit_orth(dim, dmx, m, phi, sphi, phi_m);

        GlobalC::hm.hpw.h_1psi_gpu(dim, phi_m, hphi, sphi);

        T em_host = 0;
        em_host = ddot_real(dim, phi_m, hphi);

        CHECK_CUDA(hipMemcpy(&e[m], &em_host, sizeof(T), hipMemcpyHostToDevice));

        int iter = 0;
        T gg_last = 0.0;
        T cg_norm = 0.0;
        T theta = 0.0;
        bool converged = false;
        // cg iteration

        for (iter = 0;iter < maxter; iter++)
        {
            this->calculate_gradient( precondition, dim, hphi, sphi, g, pphi );
            this->orthogonal_gradient( dim, dmx, g, scg, lagrange, phi, m );
            this->calculate_gamma_cg( iter, dim, precondition, g, scg,
			    g0, cg, gg_last, cg_norm, theta, phi_m);// scg used as sg
            converged = this->update_psi( dim, cg_norm, theta, pphi, cg, scg, phi_m ,
			    em_host, eps, hphi, sphi); // pphi is used as hcg
            hipMemcpy(&e[m], &em_host, sizeof(T), hipMemcpyHostToDevice);
            if ( converged ) break;
        }//end iter

        CHECK_CUDA(hipMemcpy(&phi[m*dmx], phi_m, dim*sizeof(T2), hipMemcpyDeviceToDevice));

        if (!converged)
        {
            ++notconv;
        }

        avg_iter += static_cast<T>(iter) + 1.00;

        if (m > 0 && reorder)
        {
			ModuleBase::GlobalFunc::NOTE("reorder bands!");
            T* e_host;
            e_host = (T*)malloc(n_band*sizeof(T));
            ModuleBase::GlobalFunc::ZEROS(e_host, n_band);
            CHECK_CUDA(hipMemcpy(e_host, e, n_band*sizeof(T), hipMemcpyDeviceToHost));

            if (e_host[m]-e_host[m-1]<-2.0*eps)
            {
                // if the last calculated eigenvalue is not the largest...
                int i=0;
                for (i=m-2; i>= 0; i--)
                {
                    if (e_host[m]-e_host[i]>2.0*eps) break;
                }
                i++;
                moved++;

                // last calculated eigenvalue should be in the i-th position: reorder
                T e0 = e_host[m];

                CHECK_CUDA(hipMemcpy(pphi, &phi[m*dmx], dim*sizeof(T2), hipMemcpyDeviceToDevice));

                for (int j = m;j >= i + 1;j--)
                {
                    e_host[j]=e_host[j-1];
                    CHECK_CUDA(hipMemcpy(&phi[j*dmx], &phi[(j-1)*dmx], dim*sizeof(T2), hipMemcpyDeviceToDevice));
                }

                e_host[i] = e0;

                CHECK_CUDA(hipMemcpy(&phi[i*dmx], pphi, dim*sizeof(T2), hipMemcpyDeviceToDevice));
                // this procedure should be good if only a few inversions occur,
                // extremely inefficient if eigenvectors are often in bad order
                // (but this should not happen)
            } // endif

            CHECK_CUDA(hipMemcpy(e, e_host, n_band*sizeof(T), hipMemcpyHostToDevice));
            delete [] e_host;
        } //end reorder

    }//end m

    avg_iter /= n_band;

    // timer::tick("Diago_CG_GPU","diag");
    CHECK_CUDA(hipFree(lagrange));
    CHECK_CUDA(hipFree(pphi));
    CHECK_CUDA(hipFree(g0));
    CHECK_CUDA(hipFree(cg));
    CHECK_CUDA(hipFree(g));
    CHECK_CUDA(hipFree(hphi));
    CHECK_CUDA(hipFree(scg));
    CHECK_CUDA(hipFree(sphi));
    CHECK_CUDA(hipFree(phi_m));

    ModuleBase::timer::tick("Diago_CG_GPU","diag");
    return;
} // end subroutine ccgdiagg


template<class T, class T2>
void Diago_CG_GPU<T, T2>::calculate_gradient(
    const T* precondition, const int dim,
    const T2 *hpsi, const T2 *spsi,
    T2 *g, T2 *ppsi)
{
    if (test_cg==1) ModuleBase::TITLE("Diago_CG_GPU","calculate_gradient");
    ModuleBase::timer::tick("Diago_CG_GPU","calculate_grad");

    int thread = 512;
    int block = (dim + thread - 1) / thread;

    // kernel_precondition(data, res, size, precondition)
    // (2) PH|psi> : g[i] = hpsi[i]/precondition[i]
    kernel_precondition<<<block, thread>>>(g, hpsi, dim, precondition);
    // (3) PS|psi> : ppsi[i] = spsi[i]/precondition[i]
    kernel_precondition<<<block, thread>>>(ppsi, spsi, dim, precondition);

    // Update lambda !
    // (4) <psi|SPH|psi >
    const T eh = this->ddot_real(dim, spsi, g);
    // (5) <psi|SPS|psi >
    const T es = this->ddot_real(dim, spsi, ppsi);
    const T lambda = eh / es;

    // Update g !
    kernel_get_gredient<<<block, thread>>>(g, ppsi, dim, lambda);
    // kernel_multi_add<<<block, thread>>>(g, g, 1, ppsi, -lambda, dim);
    ModuleBase::timer::tick("Diago_CG_GPU","calculate_grad");
    return;
}


template<class T, class T2>
void Diago_CG_GPU<T, T2>::orthogonal_gradient( const int &dim, const int &dmx,
                                    T2 *g, T2 *sg, T2 *lagrange,
                                    const T2 *eigenfunction, const int m)
{
    if (test_cg==1) ModuleBase::TITLE("Diago_CG_GPU","orthogonal_gradient");
    ModuleBase::timer::tick("Diago_CG_GPU","orth_grad");

    GlobalC::hm.hpw.s_1psi_gpu(dim, g, sg);

    int inc=1;

    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    hipblasOperation_t trans1 = HIPBLAS_OP_C;
    // ONE ZERO cufftcomplex?
    // hipblasZgemv(handle, trans1, dim, m, ONE, eigenfunction, dmx, sg, inc, ZERO, lagrange, inc);
    T2 ONE, ZERO, NEG_ONE;
    ONE.y = ZERO.x = ZERO.y = 0.0;
    ONE.x = 1.0;
    NEG_ONE.x = -1.0;
    hipblasZgemv(diag_handle, trans1, dim, m, &ONE, eigenfunction, dmx, sg, inc, &ZERO, lagrange, inc);
    /*for (int i=0; i<m; i++)
    {
        lagrange[i] = ZERO;
        for (int j=0; j<dim; j++)
        {
            lagrange[i] += conj( eigenfunction(i,j) ) * sg[j];
        }
    }*/

    // Parallel_Reduce::reduce_complex_double_pool(lagrange, m); // todo
    // (3) orthogonal |g> and |Sg> to all states (0~m-1)
    hipblasOperation_t trans2 = HIPBLAS_OP_N;

    hipblasZgemv(diag_handle, trans2, dim, m, &NEG_ONE, eigenfunction, dmx, lagrange, inc, &ONE, g, inc);
    hipblasZgemv(diag_handle, trans2, dim, m, &NEG_ONE, eigenfunction, dmx, lagrange, inc, &ONE, sg, inc);

    /*for (int i=0; i<m; i++)
    {
        for (int j=0; j<dim; j++)
        {
            const complex<T> oo = lagrange[i] * eigenfunction(i, j);
            g[j] -= oo;
            sg[j] -= oo;
        }
    }*/

    ModuleBase::timer::tick("Diago_CG_GPU","orth_grad");
    // hipblasDestroy(handle);
    return;
}

template<class T, class T2>
void Diago_CG_GPU<T, T2>::calculate_gamma_cg(
    const int iter,
    const int dim,
    const T *precondition,
    const T2 *g,
    const T2 *sg,
    T2 *psg,
    T2 *cg,
    T &gg_last,
    const T &cg_norm,
    const T &theta,
    const T2 *psi_m)
{
    if (test_cg==1) ModuleBase::TITLE("Diago_CG_GPU","calculate_gamma_cg");
    ModuleBase::timer::tick("Diago_CG_GPU","gamma_cg");
    T gg_inter;
    if (iter>0)
    {
        // (1) Update gg_inter!
        // gg_inter = <g|psg>
        // Attention : the 'g' in psg is getted last time
        gg_inter = this->ddot_real( dim, g, psg );// b means before
    }

    // (2) Update for psg!
    // two usage:
    // firstly, for now, calculate: gg_now
    // secondly, prepare for the next iteration: gg_inter
    // |psg> = P | Sg >
    // for (int i=0; i<dim; i++)
    // {
    //     psg[i] = precondition[i] * sg[i];
    // }

    int thread = 512;
    int block = (dim + thread - 1) / thread;
    kernel_precondition_inverse<<<block, thread>>>(psg, sg, dim, precondition);

    // (3) Update gg_now!
    // gg_now = < g|P|sg > = < g|psg >
    const T gg_now = this->ddot_real( dim, g, psg );

    if (iter==0)
    {
        // (40) gg_last first value : equal gg_now
        gg_last = gg_now;
        // (50) cg direction first value : |g>
        // |cg> = |g>

        // for (int i=0; i<dim; i++)
        // {
        //     cg[i] = g[i];
        // }
        CHECK_CUDA(hipMemcpy(cg, g, dim*sizeof(T2), hipMemcpyDeviceToDevice));
    }
    else
    {
        // (4) Update gamma !
        assert( gg_last != 0.0 );
        const T gamma = (gg_now - gg_inter) / gg_last;

        // (5) Update gg_last !
        gg_last = gg_now;

        // (6) Update cg direction !(need gamma and |go> ):
        // for (int i=0; i<dim; i++)
        // {
        //     cg[i] = gamma * cg[i] + g[i];
        // }

        kernel_get_gammacg<<<block, thread>>>(dim, cg, g, gamma);

        const T norma = gamma * cg_norm * sin(theta);
        // for (int i = 0;i < dim;i++)
        // {
        //     cg[i] -= norma * psi_m[i];
        // }

        kernel_get_normacg<<<block, thread>>>(dim, cg, psi_m, norma);
    }
    ModuleBase::timer::tick("Diago_CG_GPU","gamma_cg");
    return;
}


template<class T, class T2>
bool Diago_CG_GPU<T, T2>::update_psi(
    const int dim,
    T &cg_norm,
    T &theta,
    T2 *hcg,
    const T2 *cg,
    T2 *scg,
    T2 *psi_m ,
    T &eigenvalue,
    const T &threshold,
    T2 *hpsi,
    T2 *sphi)
{
    if (test_cg==1) ModuleBase::TITLE("Diago_CG_GPU","update_psi");
    ModuleBase::timer::tick("Diago_CG_GPU","update_psi");
    int thread = 512;
    int block = (dim + thread - 1) / thread;
    // pw.h_1psi(dim, cg, hcg, scg); // TODO
    // to cpu
    GlobalC::hm.hpw.h_1psi_gpu(dim, cg, hcg, scg);
    // hpsi end

    cg_norm = sqrt( this->ddot_real(dim, cg, scg) );

    if (cg_norm < 1.0e-10 ) return 1;

    const T a0 = this->ddot_real(dim, psi_m, hcg) * 2.0 / cg_norm;
    const T b0 = this->ddot_real(dim, cg, hcg) / ( cg_norm * cg_norm ) ;

    const T e0 = eigenvalue;

    theta = atan( a0/ (e0-b0) )/2.0;

    const T new_e = (e0 - b0) * cos(2.0*theta) + a0 * sin(2.0*theta);

    const T e1 = ( e0 + b0 + new_e ) /2.0;
    const T e2 = ( e0 + b0 - new_e ) /2.0;
    if (e1>e2)
    {
        theta +=  ModuleBase::PI_HALF;
    }

    eigenvalue = min( e1, e2 );

    const T cost = cos(theta);
    const T sint_norm = sin(theta)/cg_norm;

//	cout << "\n cg_norm = " << this->ddot(dim, cg, cg);
//	cout << "\n cg_norm_fac = "<< cg_norm * cg_norm;
//	cout << "\n overlap = "  << this->ddot(dim, psi_m, psi_m);

    // for (int i=0; i<dim; i++)
    // {
    //     psi_m[i] = psi_m[i] * cost + sint_norm * cg[i];
    // }

    kernel_multi_add<<<block, thread>>>(psi_m, psi_m, cost, cg, sint_norm, dim);

//	cout << "\n overlap2 = "  << this->ddot(dim, psi_m, psi_m);

    if ( abs(eigenvalue-e0)< threshold)
    {
        ModuleBase::timer::tick("Diago_CG_GPU","update_psi");
        return 1;
    }
    else
    {
        // for (int i=0; i<dim; i++)
        // {
        //     sphi[i] = sphi[i] * cost + sint_norm * scg[i];
        //     hpsi[i] = hpsi[i] * cost + sint_norm * hcg[i];
        // }
        kernel_multi_add<<<block, thread>>>(sphi, sphi, cost, scg, sint_norm, dim);
        kernel_multi_add<<<block, thread>>>(hpsi, hpsi, cost, hcg, sint_norm, dim);
        ModuleBase::timer::tick("Diago_CG_GPU","update_psi");
        return 0;
    }
}

template<class T, class T2>
void Diago_CG_GPU<T, T2>::schmit_orth
(
    const int& dim,
    const int& dmx,
    const int& m,     //end
    const T2 *psi, // matrix
    T2 *sphi,
    T2 *psi_m
)
{
    ModuleBase::timer::tick("Diago_CG_GPU","schmit_orth");
    assert( m >= 0 );
    // cout<<"orth, dim="<<dim<<endl;

    T2 *lagrange;
    CHECK_CUDA(hipMalloc((void**)&lagrange, (m+1)*sizeof(T2)));
    int inc=1;
    int mp1 = m+1;

    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    hipblasOperation_t trans1 = HIPBLAS_OP_C;

    T2 ONE, ZERO, NEG_ONE;
    ONE.y = ZERO.x = ZERO.y = 0.0;
    ONE.x = 1.0;
    NEG_ONE.x = -1.0;
    hipblasZgemv(diag_handle, trans1, dim, mp1, &ONE, psi, dmx, sphi, inc, &ZERO, lagrange, inc);

    T psi_norm;
    CHECK_CUDA(hipMemcpy(&psi_norm, &lagrange[m], sizeof(T), hipMemcpyDeviceToHost));
    hipblasOperation_t trans2 = HIPBLAS_OP_N;
    hipblasZgemv(diag_handle, trans2, dim, m, &NEG_ONE, psi, dmx, lagrange, inc, &ONE, psi_m, inc);

    psi_norm -= ddot_real(m, lagrange, lagrange); //next
    psi_norm = sqrt(psi_norm);

    int thread = 512;
    int block = (dim + thread - 1) / thread;
    kernel_normalization<<<block, thread>>>(psi_m, dim, psi_norm);

    GlobalC::hm.hpw.s_1psi_gpu(dim, psi_m, sphi);

    // hipblasDestroy(handle);
    ModuleBase::timer::tick("Diago_CG_GPU","schmit_orth");
    CHECK_CUDA(hipFree(lagrange));
    return ;
}


template<class T, class T2>
T Diago_CG_GPU<T, T2>::ddot_real
(
    const int &dim,
    const T2* psi_L,
    const T2* psi_R,
    const bool reduce
)
{
    int dim2=2*dim;
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    T result;
    // todo:
    hipblasDdot(diag_handle, dim2, (T*)psi_L, 1, (T*)psi_R, 1, &result);
    // hipblasDestroy(handle);
    return result;
}

template<class T, class T2>
T2 Diago_CG_GPU<T, T2>::ddot
(
    const int & dim,
    const T2 * psi_L,
    const T2 * psi_R
)
{
    // for (int i = 0; i < dim ; i++)
    // {
    //     result += conj(psi_L[i]) *  psi_R[i] ;
    // }
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    T2 result;
    hipblasZdotc(diag_handle, dim, psi_L, 1, psi_R, 1, &result);
    // Parallel_Reduce::reduce_complex_double_pool( result );
    // hipblasDestroy(handle);
    return result;
}  // end of ddot

// this return <psi(m)|psik>
template<class T, class T2>
T2 Diago_CG_GPU<T, T2>::ddot
(
    const int & dim,
    const T2 *psi, //complex
    const int & m,
    T2 *psik
)
{
    // assert(dim > 0) ;
    // for (int i = 0; i < dim ; i++)
    // {
    //     result += conj(psi(m, i)) *  psik[i] ;
    // }
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    T2 result;
    hipblasZdotc(diag_handle, dim, &psi[m*dim], 1, psik, 1, &result);
    // Parallel_Reduce::reduce_complex_double_pool( result );
    // hipblasDestroy(handle);
    return result;
}  // end of ddot


// this return <psi_L(m) | psi_R(n)>
template<class T, class T2>
T2 Diago_CG_GPU<T, T2>::ddot
(
    const int & dim,
    const T2 *psi_L,
    const int & m,
    const T2 *psi_R,
    const int & n
)
{
    // assert( (dim>0) && (dim<=psi_L.nc) && (dim<=psi_R.nc) );

    // for ( int i = 0; i < dim ; i++)
    // {
    //     result += conj( psi_L(m,i) ) * psi_R(n,i) ;
    // }
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    T2 result;
    hipblasZdotc(diag_handle, dim, &psi_L[m*dim], 1, &psi_R[n*dim], 1, &result);
    // Parallel_Reduce::reduce_complex_double_pool( result );

    // hipblasDestroy(handle);
    return result;
} // end of ddot

template class Diago_CG_GPU<double, double2>;
// template class Diago_CG_GPU<float, float2>;
