#include "hip/hip_runtime.h"
#include "use_fft.h"
#include "global.h"
#include "hipfft/hipfft.h"

__global__ void kernel_set(int size, CUFFT_COMPLEX *dst, const CUFFT_COMPLEX *src, const int *index_list)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int p = index_list[idx];
    if(idx < size)
    {
        dst[p].x = src[idx].x;
        dst[p].y = src[idx].y;
    }
}

__global__ void kernel_roundtrip(int size, CUFFT_COMPLEX *dst, const double *src)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x *= src[idx];
        dst[idx].y *= src[idx];
    }
}

__global__ void kernel_normalization(int size, CUFFT_COMPLEX *data, double norm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

__global__ void kernel_reorder(CUFFT_COMPLEX *dst, CUFFT_COMPLEX *src, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x = src[idx].x;
        dst[idx].y = src[idx].y;
    }
}

// Donghs fix 2021.9.8
// void Reorder_psi_plus(CUFFT_COMPLEX *dst, CUFFT_COMPLEX *src)
// {
//     ModuleBase::timer::tick("Use_FFT","reorder_psi_plus");
//     int ii = 0;
//     int size_z = GlobalC::pw.FFT_wfc.npps[0];
//     int thread = 512;
//     int block = (size_z + thread - 1) / thread;
//     for(int is=0; is<GlobalC::pw.FFT_wfc.nst; is++)
//     {
//         int ir = GlobalC::pw.FFT_wfc.ismap[is];
//         kernel_reorder<<<block, thread>>>(&dst[ir*size_z], &src[ii*size_z], size_z);
//         // CHECK_CUDA(hipMemcpy(&dst[ir*size_z], &src[ii*size_z], size_z*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToDevice));
//         ii++;
//     }
//     ModuleBase::timer::tick("Use_FFT","reorder_psi_plus");
// }

// void Reorder_psi_minus(CUFFT_COMPLEX *dst, CUFFT_COMPLEX *src)
// {
//     ModuleBase::timer::tick("Use_FFT","reorder_psi_minus");
//     int ii = 0;
//     int size_z = GlobalC::pw.FFT_wfc.npps[0];
//     int thread = 512;
//     int block = (size_z + thread - 1) / thread;
//     for(int j=0; j<GlobalC::pw.FFT_wfc.nst; j++)
//     {
//         int ir = GlobalC::pw.FFT_wfc.ismap[j];
//         kernel_reorder<<<block, thread>>>(&dst[ii*size_z], &src[ir*size_z], size_z);
//         // CHECK_CUDA(hipMemcpy(&dst[ii*size_z], &src[ir*size_z], size_z*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToDevice));
//         ii++;
//     }
//     ModuleBase::timer::tick("Use_FFT","reorder_psi_minus");
// }


void RoundTrip_kernel(const CUFFT_COMPLEX *psi, const double *vr, const int *fft_index, CUFFT_COMPLEX *psic)
{
    // (1) set value
    int thread = 512;
    int block = (GlobalC::wf.npw + thread - 1) / thread;
    int block2 = (GlobalC::pw.nrxx + thread - 1) / thread;
    kernel_set<<<block, thread>>>(GlobalC::wf.npw, psic, psi, fft_index);

    // CUFFT_COMPLEX *ordered_psi;
    // CHECK_CUDA(hipMalloc((void**)&ordered_psi, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX)));
    // CHECK_CUDA(hipMemset(ordered_psi, 0, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX)));

    // Reorder_psi_plus(ordered_psi, psic);

    // hipfftHandle cufftplan_gpu;
    // hipfftPlan3d(&cufftplan_gpu, GlobalC::pw.nx, GlobalC::pw.ny, GlobalC::pw.nz, HIPFFT_Z2Z);
    hipfftExecZ2Z(GlobalC::UFFT.fft_handle, psic, psic, HIPFFT_BACKWARD);
    // hipfftDestroy(cufftplan_gpu);

    // int block3 = (GlobalC::pw.nrxx + thread - 1) / thread;
    // kernel_normalization<<<block3, thread>>>(GlobalC::pw.nrxx, psic, (double)(GlobalC::pw.nrxx));

    kernel_roundtrip<<<block2, thread>>>(GlobalC::pw.nrxx, psic, vr);

    // hipfftHandle cufftplan_gpu2;
    // hipfftPlan3d(&cufftplan_gpu, GlobalC::pw.nx, GlobalC::pw.ny, GlobalC::pw.nz, HIPFFT_Z2Z);
    hipfftExecZ2Z(GlobalC::UFFT.fft_handle, psic, psic, HIPFFT_FORWARD);
    // hipfftDestroy(cufftplan_gpu);

    // Reorder_psi_minus(psic, ordered_psi);

    int block3 = (GlobalC::pw.nrxx + thread - 1) / thread;
    kernel_normalization<<<block3, thread>>>(GlobalC::pw.nrxx, psic, (double)(GlobalC::pw.nrxx));

    // CHECK_CUDA(hipFree(ordered_psi));

    return;
}